#include "hip/hip_runtime.h"
#ifndef __DETECT__CU__
#define __DETECT__CU__
#include <algorithm>
#include "gPET.h"
#include "externCUDA.h"
#include <thrust/device_ptr.h>
#include <thrust/sort.h>

compare_parn compare1;
compare_siten compare2;
compare_t compare3;

using namespace std;

struct object_t InitializeObject()
{
    struct object_t q;

    q.panel = 0;

    q.lengthx=0.0f;
    q.lengthy=0.0f;
    q.lengthz=0.0f;

    q.MODx=0.0f;
    q.MODy=0.0f;
    q.MODz=0.0f;

    q.Mspacex=0.0f;
    q.Mspacey=0.0f;
    q.Mspacez=0.0f;

    q.LSOx=0.0f;
    q.LSOy=0.0f;
    q.LSOz=0.0f;

    q.spacex=0.0f;
    q.spacey=0.0f;
    q.spacez=0.0f;

    q.offsetx=0.0f;
    q.offsety=0.0f;
    q.offsetz=0.0f;

    q.directionx=0.0f;
    q.directiony=0.0f;
    q.directionz=0.0f;

    q.UniXx=0.0f;
    q.UniXy=0.0f;
    q.UniXz=0.0f;

    q.UniYx=0.0f;
    q.UniYy=0.0f;
    q.UniYz=0.0f;

    q.UniZx=0.0f;
    q.UniZy=0.0f;
    q.UniZz=0.0f;

    return q;
}

void read_file_ro(struct object_t** objectArray, struct object_v** objectMaterial, int* total_Panels, char fname[100])
/********************************************************************************
c* read geometry files using rotational definition                              *
c* Input:                                                                       *
c*  fname: input geometry file                                                  *
c* Output:                                                                      *
    objectArray: buildup geometry                                               *
c*  total_Panels: total panel numbers                                           *
/*******************************************************************************/
{
    printf("loading PET detector geometry parameters ... %s\n",fname);

    FILE* fp=fopen(fname,"r");
    char buffer[256];
    int count = 0;
    fgets(buffer, 256, fp);
    fscanf(fp, "%d \n", &count);
    *total_Panels = count;
    cout << "total panels "<<*total_Panels << endl;

    float rot[3];
    fgets(buffer, 256, fp);
    fscanf(fp, "%f %f %f\n", &rot[0], &rot[1], &rot[2]);
    cout << "panel rotational axis "<<rot[0] <<" "<<rot[1] <<" "<<rot[2]<< endl;

    float rotAng;
    fgets(buffer, 256, fp);
    fscanf(fp, "%f\n", &rotAng);
    cout << "panel rotational angle "<<rotAng<< endl;


    // read the file for the second time, to load all the parameters
    struct object_t* temp;
    temp = (object_t*)malloc(*total_Panels*sizeof(object_t));
    struct object_v* temp1;
    temp1 = (object_v*)malloc(2*sizeof(object_v));
    for (int i = 0; i < *total_Panels; i++)
    {
        temp[i] = InitializeObject();
    }

    int mat = 0, pane = 0;
    float den = 0.0f, lenx=0.0f, leny=0.0f, lenz=0.0f, Mx=0.0f,My=0.0f,Mz=0.0f, Msx=0.0f,Msy=0.0f,Msz=0.0f;
    float Lx=0.0f,Ly=0.0f,Lz=0.0f,sx=0.0f, sy=0.0f, sz=0.0f, ox=0.0f, oy=0.0f, oz=0.0f, dx=0.0f,dy=0.0f,dz=0.0f;
    float UXx=0.0f, UXy=0.0f, UXz=0.0f, UYx=0.0f, UYy=0.0f, UYz=0.0f, UZx=0.0f, UZy=0.0f, UZz=0.0f;

// only two materials (0:LSO, 1: air)
    fgets(buffer, 256, fp);
    for (int i = 0; i < 2; i++)
    {
        fscanf(fp, "%d %f \n", &mat, &den);
        temp1[i].material=mat;
        temp1[i].density=den;
        //printf("mat=%d, den=%f\n", mat, den);
    }
    fgets(buffer, 256, fp);

// read in parameter for each Panel
    for (int i = 0; i < 1; i++)
    {
        //printf("i=%d\n", i);

        fgets(buffer, 256, fp);
        fscanf(fp, "%d \n", &pane);
        temp[i].panel = pane;
        //printf("starting panel=%d\n", pane);

        fgets(buffer, 256, fp);
        fscanf(fp, "%f %f %f\n", &lenx, &leny, &lenz);
        temp[i].lengthx = lenx;
        temp[i].lengthy = leny;
        temp[i].lengthz = lenz;
        //printf("lengthx=%f, lengthy=%f, lengthz=%f\n", lenx, leny, lenz);

        fgets(buffer, 256, fp);
        fscanf(fp, "%f %f %f\n", &Mx, &My, &Mz);
        temp[i].MODx = Mx;
        temp[i].MODy = My;
        temp[i].MODz = Mz;
        //printf("MODx=%f, MODy=%f, MODz=%f\n", Mx, My, Mz);

        fgets(buffer, 256, fp);
        fscanf(fp, "%f %f %f\n", &Msx, &Msy, &Msz);
        temp[i].Mspacex = Msx;
        temp[i].Mspacey = Msy;
        temp[i].Mspacez = Msz;
        //printf("Mspacex=%f, Mspacey=%f, Mspacez=%f\n", Msx, Msy, Msz);

        fgets(buffer, 256, fp);
        fscanf(fp, "%f %f %f\n", &Lx, &Ly, &Lz);
        temp[i].LSOx = Lx;
        temp[i].LSOy = Ly;
        temp[i].LSOz = Lz;
        //printf("LSOx=%f, LSOy=%f, LSOz=%f\n", Lx, Ly, Lz);

        fgets(buffer, 256, fp);
        fscanf(fp, "%f %f %f\n", &sx, &sy, &sz);
        temp[i].spacex = sx;
        temp[i].spacey = sy;
        temp[i].spacez = sz;
        //printf("spacex=%f, spacey=%f, spacez=%f\n", sx, sy, sz);



        fgets(buffer, 256, fp);
        fscanf(fp, "%f %f %f\n", &dx, &dy, &dz);
        temp[i].directionx = dx;
        temp[i].directiony = dy;
        temp[i].directionz = dz;
        //printf("directionx=%f, directiony=%f, directionz=%f\n", dx, dy, dz);



        fgets(buffer, 256, fp);
        fscanf(fp, "%f %f %f\n", &ox, &oy, &oz);
        temp[i].offsetx = ox;
        temp[i].offsety = oy;
        temp[i].offsetz = oz;
        //printf("offsetx=%f, offsety=%f, offsetz=%f\n", ox, oy, oz);


        fgets(buffer, 256, fp);
        fscanf(fp, "%f %f %f\n", &UXx, &UXy, &UXz);
        temp[i].UniXx = UXx;
        temp[i].UniXy = UXy;
        temp[i].UniXz = UXz;
        //printf("UniXx=%f, UniXy=%f, UniXz=%f\n", UXx, UXy, UXz);

        fgets(buffer, 256, fp);
        fscanf(fp, "%f %f %f\n", &UYx, &UYy, &UYz);
        temp[i].UniYx = UYx;
        temp[i].UniYy = UYy;
        temp[i].UniYz = UYz;
        //printf("UniYx=%f, UniYy=%f, UniYz=%f\n", UYx, UYy, UYz);

        fgets(buffer, 256, fp);
        fscanf(fp, "%f %f %f\n", &UZx, &UZy, &UZz);
        temp[i].UniZx = UZx;
        temp[i].UniZy = UZy;
        temp[i].UniZz = UZz;
        //printf("UniZx=%f, UniZy=%f, UniZz=%f\n", UZx, UZy, UZz);

        fgets(buffer, 256, fp);
    }
    fclose(fp);

    float vec[3];
    float ang;
    for (int i = 1; i < *total_Panels; i++)
    {
        if (*total_Panels<=1)
            break;
        //printf("i=%d\n", i);
        temp[i].panel = i;
        //printf("panel=%d\n", temp[i].panel);

        temp[i].lengthx = temp[0].lengthx;
        temp[i].lengthy = temp[0].lengthy;
        temp[i].lengthz = temp[0].lengthz;
        //printf("lengthx=%f, lengthy=%f, lengthz=%f\n", temp[i].lengthx, temp[i].lengthy, temp[i].lengthz);

        temp[i].MODx = temp[0].MODx;
        temp[i].MODy = temp[0].MODy;
        temp[i].MODz = temp[0].MODz;

        temp[i].Mspacex = temp[0].Mspacex;
        temp[i].Mspacey = temp[0].Mspacey;
        temp[i].Mspacez = temp[0].Mspacez;

        temp[i].LSOx = temp[0].LSOx;
        temp[i].LSOy = temp[0].LSOy;
        temp[i].LSOz = temp[0].LSOz;

        temp[i].spacex = temp[0].spacex;
        temp[i].spacey = temp[0].spacey;
        temp[i].spacez = temp[0].spacez;

        temp[i].directionx = temp[0].directionx;
        temp[i].directiony = temp[0].directiony;
        temp[i].directionz = temp[0].directionz;


        ang=rotAng*PI/180.0f*i;

        vec[0] = temp[0].offsetx;
        vec[1] = temp[0].offsety;
        vec[2] = temp[0].offsetz;
        temp[i].offsetx=(1-cosf(ang))*(vec[0]*rot[0])*rot[0]+cosf(ang)*vec[0]+sinf(ang)*(rot[1]*vec[2]-rot[2]*vec[1]);
        temp[i].offsety=(1-cosf(ang))*(vec[1]*rot[1])*rot[1]+cosf(ang)*vec[1]+sinf(ang)*(rot[2]*vec[0]-rot[0]*vec[2]);
        temp[i].offsetz=(1-cosf(ang))*(vec[2]*rot[2])*rot[2]+cosf(ang)*vec[2]+sinf(ang)*(rot[0]*vec[1]-rot[1]*vec[0]);
        //printf("offsetx=%f, offsety=%f, offsetz=%f\n", temp[i].offsetx, temp[i].offsety, temp[i].offsetz);

        vec[0] = temp[0].UniXx;
        vec[1] = temp[0].UniXy;
        vec[2] = temp[0].UniXz;
        temp[i].UniXx = (1-cosf(ang))*(vec[0]*rot[0])*rot[0]+cosf(ang)*vec[0]+sinf(ang)*(rot[1]*vec[2]-rot[2]*vec[1]);
        temp[i].UniXy = (1-cosf(ang))*(vec[1]*rot[1])*rot[1]+cosf(ang)*vec[1]+sinf(ang)*(rot[2]*vec[0]-rot[0]*vec[2]);
        temp[i].UniXz = (1-cosf(ang))*(vec[2]*rot[2])*rot[2]+cosf(ang)*vec[2]+sinf(ang)*(rot[0]*vec[1]-rot[1]*vec[0]);
        //printf("UniXx=%f, UniXy=%f, UniXz=%f\n", temp[i].UniXx, temp[i].UniXy, temp[i].UniXz);

        vec[0] = temp[0].UniYx;
        vec[1] = temp[0].UniYy;
        vec[2] = temp[0].UniYz;
        temp[i].UniYx = (1-cosf(ang))*(vec[0]*rot[0])*rot[0]+cosf(ang)*vec[0]+sinf(ang)*(rot[1]*vec[2]-rot[2]*vec[1]);
        temp[i].UniYy = (1-cosf(ang))*(vec[1]*rot[1])*rot[1]+cosf(ang)*vec[1]+sinf(ang)*(rot[2]*vec[0]-rot[0]*vec[2]);
        temp[i].UniYz = (1-cosf(ang))*(vec[2]*rot[2])*rot[2]+cosf(ang)*vec[2]+sinf(ang)*(rot[0]*vec[1]-rot[1]*vec[0]);
        //printf("UniYx=%f, UniYy=%f, UniYz=%f\n", temp[i].UniYx, temp[i].UniYy, temp[i].UniYz);

        vec[0] = temp[0].UniZx;
        vec[1] = temp[0].UniZy;
        vec[2] = temp[0].UniZz;
        temp[i].UniZx = (1-cosf(ang))*(vec[0]*rot[0])*rot[0]+cosf(ang)*vec[0]+sinf(ang)*(rot[1]*vec[2]-rot[2]*vec[1]);
        temp[i].UniZy = (1-cosf(ang))*(vec[1]*rot[1])*rot[1]+cosf(ang)*vec[1]+sinf(ang)*(rot[2]*vec[0]-rot[0]*vec[2]);
        temp[i].UniZz = (1-cosf(ang))*(vec[2]*rot[2])*rot[2]+cosf(ang)*vec[2]+sinf(ang)*(rot[0]*vec[1]-rot[1]*vec[0]);
        //printf("UniZx=%f, UniZy=%f, UniZz=%f\n", temp[i].UniZx, temp[i].UniZy, temp[i].UniZz);
    }
    *objectArray = temp;
    *objectMaterial = temp1;
    
    //printf("\n");
    printf("\n");
}

int outevents(int* num_d, Event* totalevents_d, const char *outputfilename)
{
//copy data from device to host
//renewed at 1025, do not have to be the memory on GPU
    int num;
    if(hipMemcpy(&num, num_d, sizeof(int), hipMemcpyDeviceToHost)!=hipSuccess)
        num=*num_d;
    cout<<"num is "<<num<<endl;
    if(num <1) return 0;
    Event* tempData_h =(struct Event*) malloc( sizeof(Event)*num);
    if(hipMemcpy(tempData_h, totalevents_d, sizeof(Event)*num, hipMemcpyDeviceToHost)!=hipSuccess)
        memcpy(tempData_h, totalevents_d, sizeof(Event)*num);
//  write results to file
    ofstream out(outputfilename,ios::app|ios::binary);
    out.write((char*) tempData_h,sizeof(Event)*num);
    out.close();
    cout<<"data has been written to "<<outputfilename<<"\n";
//  free space
    free(tempData_h);
    return 1;
}

void quicksort(Event*  hits,int start, int stop, int sorttype)
{   
    //CPU sort function for ordering events in cpu memory
    //sorttype  1 for ordering by particle #,2 for site number, 3 for flight time
    switch(sorttype)
    {
    case 1:
    {   sort(hits+start,hits+stop,compare1);
        break;
    }
    case 2:
    {   sort(hits+start,hits+stop,compare2);
        break;
    }
    case 3:
    {   sort(hits+start,hits+stop,compare3);
        break;
    }
    }
}
void quicksort_d(Event* events_d,int start, int stop, int sorttype)
{
    //GPU version for ordering the events in gpu memory,
    //more suitable for large scale sorting
    //sorttype  1 for ordering by particle #,2 for site number, 3 for flight time

    printf("gpu sort starts!!\n");
    thrust::device_ptr<Event> hits=thrust::device_pointer_cast(events_d);
    switch(sorttype)
    {
        case 1:
        {   thrust::sort(hits+start,hits+stop,compare1);
            break;
        }
        case 2:
        {   thrust::sort(hits+start,hits+stop,compare2);
            break;
        }
        case 3:
        {   thrust::sort(hits+start,hits+stop,compare3);
            break;
        }
    }//*/
    printf("gpu sort finishs!!\n");//*/
}
void quicksort_h(Event* events_d,int start, int stop, int sorttype)
{
    //CPU version for ordering the events in gpu memory,
    //more suitable for large scale sorting
    //sorttype  1 for ordering by particle #,2 for site number, 3 for flight time
    printf("cpu sort starts!!\n");
    int counts = stop-start;
    Event* events=(Event*) malloc(sizeof(Event)*counts);
    hipMemcpy(events,events_d+start,sizeof(Event)*counts,hipMemcpyDeviceToHost);
    quicksort(events,0,counts,sorttype);
    hipMemcpy(events_d+start,events,sizeof(Event)*counts,hipMemcpyHostToDevice);
    free(events);
    printf("cpu sort finishs!!\n");//*/
}

void orderevents(int* counts,Event* events_d)
{
    Event* events=(Event*) malloc(sizeof(Event)*counts[0]);
    hipMemcpy(events,events_d,sizeof(Event)*counts[0],hipMemcpyDeviceToHost);
    quicksort(events,0,counts[0],2);
    int start=0;
    for(int i=1; i<counts[0];)
    {
        while(events[i].siten==events[start].siten&&(i<counts[0]))
            i++;
        if(i>start+1) quicksort(events,start,i,3);
        start=i;
        i++;
    }
    hipMemcpy(events_d,events,sizeof(Event)*counts[0],hipMemcpyHostToDevice);
    free(events);//*/
}

void outputData(void *src, const int size, const char *outputfilename, const char *mode)
//      output data to file
{
//      copy data from GPU to CPU
    void *tempData_h = malloc( size );
    if(hipMemcpy( tempData_h, src, size, hipMemcpyDeviceToHost)!= hipSuccess) 
        memcpy(tempData_h, src, size);

//      write results to file
    FILE *fp;
    fp = fopen(outputfilename, mode);
    if( fp == NULL )
    {
        cout << "Can not open file to write results.";
        exit(1);
    }
    fwrite (tempData_h, size, 1 , fp );
    fclose(fp);

//      free space
    free(tempData_h);
}

#endif