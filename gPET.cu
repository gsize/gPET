#include "hip/hip_runtime.h"
#ifndef __GPET_CU__
#define __GPET_CU__
#include "gPET.h"
#include "externCUDA.h"

float3 xbufferRepeat[NPART];
float4 vxbufferRepeat[NPART];
double h_time[NPART];
int eventidbuffer[NPART];

using namespace std;

void simulateParticle(Particle particle,int ptype_h, int total_Panels)
{
    time_t start_time, end_time;
    float time_diff;
    start_time = clock();

    double* a= new double[NPART];//used for initialization
    memset(a,0,sizeof(double)*NPART);
    int naverage = 3;//assuming 3 singles will be recorded on average per photon
    //should change accordingly if long crystals are used
    int zero=0, temptemp[2]= {0,NPART*naverage};;

    Event* events_d;//used for recording singles
    hipMalloc(&events_d,sizeof(Event)*(NPART*naverage));
    int* counts_d;
    hipMalloc(&counts_d,sizeof(int)*naverage);

    int first=0, last = particle.NParticle, nactive_h=0, curparticle=0, nsstk_h=0, npar=0;
    size_t nShared = (total_Panels+2)*sizeof(int)+(30*total_Panels+2)*sizeof(float);

    if(ptype_h==0) npar=floor(NPART/2); // leaving space for photons --> doubling the number of positrons 
    else npar=NPART;

//  loop until all particles from ps file are done
    while(curparticle<particle.NParticle)
    {
        first=curparticle;
        last = first + npar -1;
        if(last>particle.NParticle-1)
            last=particle.NParticle-1;
                        
        nactive_h = last - first + 1; // particle number in a batch

//copy data, CUDA_CALL is defined in gPET.h 
        CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_time),a,sizeof(double)*NPART,0,hipMemcpyHostToDevice));
        
        CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(x_gBrachy), &(particle.xbuffer[first]),sizeof(float3)*nactive_h, 0, hipMemcpyHostToDevice));
        CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(vx_gBrachy), &(particle.vxbuffer[first]),sizeof(float4)*nactive_h, 0, hipMemcpyHostToDevice));
        CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_eventid), &(particle.eventid[first]),sizeof(int)*nactive_h, 0, hipMemcpyHostToDevice));
        CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_time), &(particle.time[first]),sizeof(double)*nactive_h, 0, hipMemcpyHostToDevice));               

//      simulate a batch of particles
        if (ptype_h == 0 && nactive_h>0)
        {
            // put position for photon based on the PSF of positron
            setPositionForPhoton<<<NRAND/NTHREAD_PER_BLOCK_GPET, NTHREAD_PER_BLOCK_GPET>>>(nactive_h, curparticle, useprange_h);
            hipDeviceSynchronize();
            nactive_h*=2;      
        }
//output data
#if OUTPUTPSF == 1
        CUDA_CALL(hipMemcpyFromSymbol( &(xbufferRepeat[0]), x_gBrachy,sizeof(float3)*NPART, 0, hipMemcpyDeviceToHost));
        CUDA_CALL(hipMemcpyFromSymbol(&(vxbufferRepeat[0]), vx_gBrachy, sizeof(float4)*NPART, 0, hipMemcpyDeviceToHost));
        CUDA_CALL(hipMemcpyFromSymbol(&(eventidbuffer[0]), d_eventid, sizeof(int)*NPART, 0, hipMemcpyDeviceToHost));
        CUDA_CALL(hipMemcpyFromSymbol(&(h_time[0]), d_time, sizeof(double)*NPART, 0, hipMemcpyDeviceToHost));
        //printf("copy data finished\n");
        ofstream out("./output/outsource.dat",ios::app|ios::binary);
        ofstream outid("./output/idsource.dat",ios::app|ios::binary);
        ofstream outt("./output/timesource.dat",ios::app|ios::binary);
        for(int ii=0;ii<NPART;ii++)
        {
            if(h_time[ii]>0)
            {
                out.write((char*) &(xbufferRepeat[ii]),sizeof(float3));
                out.write((char*) &(vxbufferRepeat[ii]),sizeof(float4));
                outid.write((char*) &(eventidbuffer[ii]),sizeof(int));
                outt.write((char*) &(h_time[ii]),sizeof(double));
            }
            
        }//*/
        out.close();
        outid.close();
        outt.close();
#endif
//simulate transport of photons in phantom
        photon<<<NRAND/NTHREAD_PER_BLOCK_GPET, NTHREAD_PER_BLOCK_GPET>>>(NPART);
        hipDeviceSynchronize();
//output data
#if OUTPUTPSF == 2
        CUDA_CALL(hipMemcpyFromSymbol( &(xbufferRepeat[0]), x_gBrachy,sizeof(float3)*NPART, 0, hipMemcpyDeviceToHost));
        CUDA_CALL(hipMemcpyFromSymbol(&(vxbufferRepeat[0]), vx_gBrachy, sizeof(float4)*NPART, 0, hipMemcpyDeviceToHost));
        CUDA_CALL(hipMemcpyFromSymbol(&(eventidbuffer[0]), d_eventid, sizeof(int)*NPART, 0, hipMemcpyDeviceToHost));
        CUDA_CALL(hipMemcpyFromSymbol(&(h_time[0]), d_time, sizeof(double)*NPART, 0, hipMemcpyDeviceToHost));
        //printf("copy data finished\n");
        ofstream out("./output/outphantom.dat",ios::app|ios::binary);
        ofstream outid("./output/idphantom.dat",ios::app|ios::binary);
        ofstream outt("./output/timephantom.dat",ios::app|ios::binary);
        for(int ii=0;ii<NPART;ii++)
        {
            if(h_time[ii]>0)
            {
                out.write((char*) &(xbufferRepeat[ii]),sizeof(float3));
                out.write((char*) &(vxbufferRepeat[ii]),sizeof(float4));
                outid.write((char*) &(eventidbuffer[ii]),sizeof(int));
                outt.write((char*) &(h_time[ii]),sizeof(double));
            }
            
        }
        out.close();
        outid.close();
        outt.close();
#endif
//initialize
        CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(nsstk), &zero, sizeof(int), 0, hipMemcpyHostToDevice));
        CUDA_CALL(hipMemcpy(counts_d,temptemp,sizeof(int)*2,hipMemcpyHostToDevice));
//simulate transport of photons in detector       
        photonde<<<1+(NPART-1)/NTHREAD_PER_BLOCK_GPET, NTHREAD_PER_BLOCK_GPET,nShared>>>(events_d,counts_d,NPART, first,dens_panel, mat_panel, panelID, lengthx_panel, lengthy_panel, lengthz_panel,
                MODx_panel, MODy_panel, MODz_panel, Mspacex_panel, Mspacey_panel, Mspacez_panel,
                LSOx_panel, LSOy_panel, LSOz_panel, spacex_panel, spacey_panel, spacez_panel,
                offsetx_panel, offsety_panel, offsetz_panel, directionx_panel, directiony_panel, directionz_panel,
                UniXx_panel, UniXy_panel, UniXz_panel, UniYx_panel, UniYy_panel, UniYz_panel,
                UniZx_panel, UniZy_panel, UniZz_panel);
        hipDeviceSynchronize();//*/   
        CUDA_CALL(hipMemcpyFromSymbol(&nsstk_h, HIP_SYMBOL(nsstk), sizeof(int), 0, hipMemcpyDeviceToHost));
        printf("\nthere are %d Hits in this batch\n", nsstk_h/5);
//output data
#if OUTPUTHIT==1
        void *tempData;
        hipMalloc( (void **) &tempData, nsstk_h*sizeof(int));
        CUDA_CALL(hipMemcpyFromSymbol(tempData, HIP_SYMBOL(sid), nsstk_h*sizeof(int), 0,hipMemcpyDeviceToDevice));
        outputData(tempData, nsstk_h*sizeof(int), "./output/HitsID.dat", "ab");
        hipFree(tempData);

        void *tempData2;
        CUDA_CALL(hipMalloc( (void **) &tempData2, sizeof(float)*nsstk_h));
        CUDA_CALL(hipMemcpyFromSymbol(tempData2, HIP_SYMBOL(sf), sizeof(float)*nsstk_h, 0,hipMemcpyDeviceToDevice));
        outputData(tempData2,sizeof(float)*(nsstk_h), "./output/Hits.dat", "ab");
        hipFree(tempData2);
#endif
        int counts=0;
        hipMemcpy(&counts,counts_d,sizeof(int),hipMemcpyDeviceToHost);
        cout<<"counts of events after adder is "<<counts<<endl;
        //outevents(&counts,events_d,"adder.dat");

//insert proper digitizer module in the following part
//the module that can change the number of events should be followed by a sort
        //energy and spatial blur
        blur<<<NRAND/512,512>>>(counts,events_d,blurpolicy, Eref, Rref, Eslope, Sblur);
        hipDeviceSynchronize();
        //outevents(&counts,events_d,"blur.dat");

        //energy window
        energywindow<<<NRAND/512,512>>>(counts_d,events_d, counts, Eth,2000000);
        hipDeviceSynchronize();
        //quicksort_d(events_d,0,counts,3);// could try GPU sort for large NPART
        quicksort_h(events_d,0,counts,3);// if error occurs, use the CPU srt function
        hipMemcpy(&counts,counts_d,sizeof(int),hipMemcpyDeviceToHost);
        cout<<"counts of events after thresholder is "<<counts<<endl;
        //outevents(&counts,events_d,"thresholder1.dat");

        //deadtime part
        if(dlevel!=3)
        {
            setSitenum<<<counts/512+1,512>>>(counts,events_d,dlevel);
            hipDeviceSynchronize();
            printf("set site number ok\n");
        } 
        orderevents(&counts,events_d);//make events globally ordered by site number, and then ordered by flight time in each volume
        deadtime<<<NRAND/512,512>>>(counts_d,events_d, counts, dtime, dtype);
        hipDeviceSynchronize();
        cout<<"deadtime is ok\n";
        //quicksort_d(events_d,0,counts,3);
        quicksort_h(events_d,0,counts,3);
        hipMemcpy(&counts,counts_d,sizeof(int),hipMemcpyDeviceToHost);
        cout<<"counts of events after deadtime is "<<counts<<endl;
        //outevents(&counts,events_d,"./output/deadtime.dat");//*/

        energywindow<<<NRAND/512,512>>>(counts_d,events_d, counts, Ewinmin,Ewinmax);
        hipDeviceSynchronize();
        //quicksort_d(events_d,0,counts,3);
        quicksort_h(events_d,0,counts,3);
        hipMemcpy(&counts,counts_d,sizeof(int),hipMemcpyDeviceToHost);
        cout<<"counts of singles is "<<counts<<endl;
        outevents(counts_d,events_d,"./output/singles.dat");//*/

        nactive_h = 0;
        curparticle = last+1;
    }
    hipFree(events_d);
    hipFree(counts_d);

    end_time = clock();
    time_diff = ((float)end_time - (float)start_time)/CLOCKS_PER_SEC;
    printf("\n\n****************************************\n");
    printf("Simulation time: %f s.\n\n",time_diff);
    printf("****************************************\n\n\n");
}

void sampleParticle(Source source, Isotopes isotopes, float tstart, float tend, int total_Panels)
{
    float ttemp, thalf;
    for(int i=0;i<source.NSource;i++)
    {
        thalf= isotopes.halftime[source.type[i]];
        source.natom[i]= floor(source.natom[i]*exp2(-tstart/thalf));
    }
    tend-=tstart; // set tstart as new reference time point

//initialize some parameters, avoid using gloabl variable by putting in the same function
    int *d_type, *d_shape;
    unsigned int* d_natom;
    CUDA_CALL(hipMalloc((void **) &d_natom, sizeof(unsigned int)*source.NSource));
    CUDA_CALL(hipMemcpy(d_natom,source.natom,sizeof(unsigned int)*source.NSource,hipMemcpyHostToDevice));
    CUDA_CALL(hipMalloc((void **) &d_type, sizeof(int)*source.NSource));
    CUDA_CALL(hipMemcpy(d_type,source.type,sizeof(int)*source.NSource,hipMemcpyHostToDevice));
    CUDA_CALL(hipMalloc((void **) &d_shape, sizeof(int)*source.NSource));
    CUDA_CALL(hipMemcpy(d_shape,source.shape,sizeof(int)*source.NSource,hipMemcpyHostToDevice));
    float *d_shapecoeff;
    CUDA_CALL(hipMalloc((void **) &d_shapecoeff, sizeof(float)*6*source.NSource));
    CUDA_CALL(hipMemcpy(d_shapecoeff,source.shapecoeff,sizeof(float)*6*source.NSource,hipMemcpyHostToDevice));
    
    float *d_halftime, *d_decayRatio, *d_coef;
    CUDA_CALL(hipMalloc((void **) &d_halftime, sizeof(float)*isotopes.Ntype));
    CUDA_CALL(hipMemcpy(d_halftime,isotopes.halftime,sizeof(float)*isotopes.Ntype,hipMemcpyHostToDevice));
    CUDA_CALL(hipMalloc((void **) &d_decayRatio, sizeof(float)*isotopes.Ntype));
    CUDA_CALL(hipMemcpy(d_decayRatio,isotopes.decayRatio,sizeof(float)*isotopes.Ntype,hipMemcpyHostToDevice));
    CUDA_CALL(hipMalloc((void **) &d_coef, sizeof(float)*isotopes.Ntype*8));
    CUDA_CALL(hipMemcpy(d_coef,isotopes.coef,sizeof(float)*isotopes.Ntype*8,hipMemcpyHostToDevice));

    unsigned int* sumpartial= new unsigned int[source.NSource], nemitted;
    unsigned int* d_sumpartial, totalatom;
    CUDA_CALL(hipMalloc((void **) &d_sumpartial, sizeof(unsigned int)*source.NSource));
    
    printf("finish GPU memory transfer for source information\n");
    
    int enough=1;
    unsigned int preemitted = 0, curemitted=0;
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_curemitted), &curemitted,sizeof(unsigned int), 0, hipMemcpyHostToDevice));

    double* a= new double[NPART];
    memset(a,0,sizeof(double)*NPART);

    time_t start_time, end_time;
    float time_diff;
    start_time = clock();

    int naverage = 3;//assuming 3 singles will be recorded on average per photon
    //should change accordingly if long crystals are used
    Event* events_d;
    CUDA_CALL(hipMalloc(&events_d,sizeof(Event)*(NPART*naverage)));
    int* counts_d;
    CUDA_CALL(hipMalloc(&counts_d,sizeof(int)*naverage));

    int temptemp[2]= {0,NPART*naverage};
    int zero=0, nsstk_h=0;
    size_t nShared = (total_Panels+2)*sizeof(int)+(30*total_Panels+2)*sizeof(float);

    for(int epoch=0;;epoch++)
    {
        enough=1;
        
        thalf= isotopes.halftime[source.type[0]];
        sumpartial[0]=source.natom[0];
        nemitted=floor(source.natom[0]*(1-exp2(-tend/thalf)));//set initial numbers
        for(int i=1;i<source.NSource;i++)
        {
            thalf= isotopes.halftime[source.type[i]];
            sumpartial[i]=sumpartial[i-1]+source.natom[i];
            nemitted+=floor(source.natom[i]*(1-exp2(-tend/thalf)));
        }
        totalatom=sumpartial[source.NSource-1];
        printf("tstart is %f total possible number is %d possible emitted in the remaining time interval %d\n", tstart, totalatom, nemitted);
        ttemp=tend;
        if(nemitted>NPART/2)
        {
            ttemp=findT(0, tend, isotopes, source);
            //printf("searched time point is %f time interval is %f\n", tstart+ttemp,ttemp);
            enough=0;
        }
        printf("searched time point is %f time interval is %f\n", tstart+ttemp,ttemp);
        CUDA_CALL(hipMemcpy(d_sumpartial,sumpartial,sizeof(unsigned int)*source.NSource,hipMemcpyHostToDevice));

        CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_time),a,sizeof(double)*NPART,0,hipMemcpyHostToDevice));

//set position and sampling time for photons
        setPosition <<<NRAND/NTHREAD_PER_BLOCK_GPET,NTHREAD_PER_BLOCK_GPET>>> (source.NSource, totalatom, tstart, ttemp, d_natom, d_sumpartial, d_type, d_shape, d_shapecoeff, 
             d_halftime, d_decayRatio, d_coef, useprange_h);
        hipDeviceSynchronize();

        CUDA_CALL(hipMemcpyFromSymbol(&curemitted, HIP_SYMBOL(d_curemitted),sizeof(unsigned int), 0, hipMemcpyDeviceToHost));         
        printf("currently emitted photons %d\n", (curemitted - preemitted)*2);
        CUDA_CALL(hipMemcpy(source.natom,d_natom,sizeof(unsigned int)*source.NSource,hipMemcpyDeviceToHost));

#if OUTPUTPSF == 2
        CUDA_CALL(hipMemcpyFromSymbol( &(xbufferRepeat[0]), x_gBrachy,sizeof(float3)*NPART, 0, hipMemcpyDeviceToHost));
        CUDA_CALL(hipMemcpyFromSymbol(&(vxbufferRepeat[0]), vx_gBrachy, sizeof(float4)*NPART, 0, hipMemcpyDeviceToHost));
        CUDA_CALL(hipMemcpyFromSymbol(&(eventidbuffer[0]), d_eventid, sizeof(int)*NPART, 0, hipMemcpyDeviceToHost));
        CUDA_CALL(hipMemcpyFromSymbol(&(h_time[0]), d_time, sizeof(double)*NPART, 0, hipMemcpyDeviceToHost));
        //printf("copy data finished\n");
        ofstream out1("./output/outsource.dat",ios::app|ios::binary);
        ofstream outid1("./output/idsource.dat",ios::app|ios::binary);
        ofstream outt1("./output/timesource.dat",ios::app|ios::binary);
        for(int ii=0;ii<NPART;ii++)
        {
            if(h_time[ii]>0)
            {
                out1.write((char*) &(xbufferRepeat[ii]),sizeof(float3));
                out1.write((char*) &(vxbufferRepeat[ii]),sizeof(float4));
                outid1.write((char*) &(eventidbuffer[ii]),sizeof(int));
                outt1.write((char*) &(h_time[ii]),sizeof(double));
            }
            
        }
        out1.close();
        outid1.close();
        outt1.close();
#endif

//moving time slices
        tstart+=ttemp;
        tend-=ttemp;
        
        photon<<<NRAND/512, 512>>>(NPART);
        hipDeviceSynchronize();

#if OUTPUTPSF == 2
        CUDA_CALL(hipMemcpyFromSymbol( &(xbufferRepeat[0]), x_gBrachy,sizeof(float3)*NPART, 0, hipMemcpyDeviceToHost));
        CUDA_CALL(hipMemcpyFromSymbol(&(vxbufferRepeat[0]), vx_gBrachy, sizeof(float4)*NPART, 0, hipMemcpyDeviceToHost));
        CUDA_CALL(hipMemcpyFromSymbol(&(eventidbuffer[0]), d_eventid, sizeof(int)*NPART, 0, hipMemcpyDeviceToHost));
        CUDA_CALL(hipMemcpyFromSymbol(&(h_time[0]), d_time, sizeof(double)*NPART, 0, hipMemcpyDeviceToHost));
        //printf("copy data finished\n");
        ofstream out("./output/outphantom.dat",ios::app|ios::binary);
        ofstream outid("./output/idphantom.dat",ios::app|ios::binary);
        ofstream outt("./output/timephantom.dat",ios::app|ios::binary);
        for(int ii=0;ii<NPART;ii++)
        {
            if(h_time[ii]>0)
            {
                out.write((char*) &(xbufferRepeat[ii]),sizeof(float3));
                out.write((char*) &(vxbufferRepeat[ii]),sizeof(float4));
                outid.write((char*) &(eventidbuffer[ii]),sizeof(int));
                outt.write((char*) &(h_time[ii]),sizeof(double));
            }
            
        }
        out.close();
        outid.close();
        outt.close();
#endif       

        CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(nsstk), &zero, sizeof(int), 0, hipMemcpyHostToDevice));
        CUDA_CALL(hipMemcpy(counts_d,temptemp,sizeof(int)*2,hipMemcpyHostToDevice));
        
        photonde<<<1+(NPART-1)/NTHREAD_PER_BLOCK_GPET, NTHREAD_PER_BLOCK_GPET,nShared>>>(events_d,counts_d,NPART, preemitted,dens_panel, mat_panel, panelID, lengthx_panel, lengthy_panel, lengthz_panel,
                MODx_panel, MODy_panel, MODz_panel, Mspacex_panel, Mspacey_panel, Mspacez_panel,
                LSOx_panel, LSOy_panel, LSOz_panel, spacex_panel, spacey_panel, spacez_panel,
                offsetx_panel, offsety_panel, offsetz_panel, directionx_panel, directiony_panel, directionz_panel,
                UniXx_panel, UniXy_panel, UniXz_panel, UniYx_panel, UniYy_panel, UniYz_panel,
                UniZx_panel, UniZy_panel, UniZz_panel);
        hipDeviceSynchronize();//*/   
        CUDA_CALL(hipMemcpyFromSymbol(&nsstk_h, HIP_SYMBOL(nsstk), sizeof(int), 0, hipMemcpyDeviceToHost));
        printf("\nthere are %d Hits in this batch\n", nsstk_h/5);
        preemitted = curemitted;

#if OUTPUTHIT==1
        void *tempData;
        hipMalloc( (void **) &tempData, nsstk_h*sizeof(int));
        CUDA_CALL(hipMemcpyFromSymbol(tempData, HIP_SYMBOL(sid), nsstk_h*sizeof(int), 0,hipMemcpyDeviceToDevice));
        outputData(tempData, nsstk_h*sizeof(int), "./output/HitsID.dat", "ab");
        hipFree(tempData);

        void *tempData2;
        CUDA_CALL(hipMalloc( (void **) &tempData2, sizeof(float)*nsstk_h));
        CUDA_CALL(hipMemcpyFromSymbol(tempData2, HIP_SYMBOL(sf), sizeof(float)*nsstk_h, 0,hipMemcpyDeviceToDevice));
        outputData(tempData2,sizeof(float)*(nsstk_h), "./output/Hits.dat", "ab");
        hipFree(tempData2);
#endif
        int counts=0;
        hipMemcpy(&counts,counts_d,sizeof(int),hipMemcpyDeviceToHost);
        cout<<"counts of events after adder is "<<counts<<endl;
        outevents(&counts,events_d,"./output/adder.dat");

//insert proper digitizer module in the following part
//the module that can change the number of events should be followed by a sort
        //energy blur and spatial blur
        blur<<<NRAND/512,512>>>(counts,events_d,blurpolicy, Eref, Rref, Eslope, Sblur);
        hipDeviceSynchronize();
        //outevents(&counts,events_d,"./output/blur.dat");

        //energy window
        energywindow<<<NRAND/512,512>>>(counts_d,events_d, counts, Eth,2000000);
        hipDeviceSynchronize();
        //quicksort_d(events_d,0,counts,3);
        quicksort_h(events_d,0,counts,3);
        hipMemcpy(&counts,counts_d,sizeof(int),hipMemcpyDeviceToHost);
        cout<<"counts of events after thresholder is "<<counts<<endl;
        //outevents(&counts,events_d,"./output/thresholder1.dat");

        //deadtime part
        if(dlevel!=3)
        {
            setSitenum<<<NRAND/512,512>>>(counts,events_d,dlevel);
            hipDeviceSynchronize();
            printf("set site number ok\n");
        } 
        orderevents(&counts,events_d);//make events globally ordered by site number, and then ordered by flight time in each volume
        deadtime<<<NRAND/512,512>>>(counts_d,events_d, counts, dtime, dtype);
        hipDeviceSynchronize();
        cout<<"deadtime is ok\n";
        //quicksort_d(events_d,0,counts,3);
        quicksort_h(events_d,0,counts,3);
        hipMemcpy(&counts,counts_d,sizeof(int),hipMemcpyDeviceToHost);
        cout<<"counts of events after deadtime is "<<counts<<endl;
        //outevents(&counts,events_d,"./output/deadtime.dat");//*/

        energywindow<<<NRAND/512,512>>>(counts_d,events_d, counts, Ewinmin,Ewinmax);
        hipDeviceSynchronize();
        //quicksort_d(events_d,0,counts,3);
        quicksort_h(events_d,0,counts,3);
        hipMemcpy(&counts,counts_d,sizeof(int),hipMemcpyDeviceToHost);
        cout<<"counts of singles is "<<counts<<endl;
        outevents(counts_d,events_d,"./output/singles.dat");

        printf("finish %d run\n", epoch);
        if(enough ) break;
    }
    hipFree(events_d);
    hipFree(counts_d);

    end_time = clock();
    time_diff = ((float)end_time - (float)start_time)/CLOCKS_PER_SEC;
    printf("\n\n****************************************\n");
    printf("Simulation time: %f s.\n\n",time_diff);
    printf("****************************************\n\n\n");
}

float findT(float tstart, float tend, Isotopes isotopes, Source source)
{
    float tmid= (tstart+tend)*0.5;
    float thalf;
    int nemitted=0;
    for(int i=0;i<source.NSource;i++)
    {
        thalf= isotopes.halftime[source.type[i]];
        nemitted += floor(source.natom[i]*(1-exp2(-tmid/thalf)));
    }
    if(nemitted>0.98*NPART*0.5) tmid= findT(tstart,  tmid, isotopes, source);
    else if(nemitted<0.95*NPART*0.5) tmid= findT(tmid, tend, isotopes, source);
    else return tmid;
}

#endif
